
#include <hip/hip_runtime.h>
__global__ void add2( double * v1, const double * v2 )
{
    int idx = threadIdx.x;
    v1[idx] += v2[idx];
}

__global__ void multiply1(const double * v1, const double * v2, const double * v3, const double * v4, double * v5)
{
    
    int idx = threadIdx.x;
    v5[idx] += v4[idx]*v3[idx]*v2[idx]*v1[idx]; 

}

__global__ void matrixMultiply(const double* v1, const double* v2, double* v3, int M) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
   

    if ((row < M) && (col < M)) {
        float temp = 0;
        for (int k = 0; k < M; k++) {
            temp += v1[row*M + k] * v2[k*M + col];
        }
        v3[row*M + col] = temp;
    }
}