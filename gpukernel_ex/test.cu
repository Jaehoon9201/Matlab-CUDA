
#include <hip/hip_runtime.h>
__global__ void add2( double * v1, const double * v2 )
{
    int idx = threadIdx.x;
    v1[idx] += v2[idx];
}